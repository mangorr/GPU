// ssh nyuID@access.cims.nyu.edu
// ssh cuda4
// nvcc -o vectorprog vectors.cu -lm
// password *******

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define RANGE 11.79

/*** TODO: insert the declaration of the kernel function below this line ***/
__global__ void vecGPU (float *ad, float *bd, float *cd, int n, int threads);

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++)
		temp[i] += a[i] * b[i];
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
		
	/* TODO: in this part you need to do the following:
		1. allocate ad, bd, and cd in the device
		2. send a, b, and c to the device
		3. write the kernel, call it: vecGPU
		4. call the kernel (the kernel itself will be written at the comment at the end of this file), 
		   you need to decide about the number of threads, blocks, etc and their geometry.
		5. bring the cd array back from the device and store it in c array (declared earlier in main)
		6. free ad, bd, and cd
	*/

	start = clock();

	// Transfer a and b to device memory
	int size = n * sizeof(float);
	hipMalloc((void **)&ad, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&bd, size);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	
	// Allocate device memory for cd
	hipMalloc((void **)&cd, size);
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);

	// Kernel invocation code

	int threads = 8 * 500; // Change for different values
	vecGPU<<<8,500>>>(ad,bd,cd,n, threads);

	// Transfer C from device to host
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);

	// Free device memory for A, B, C
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	
	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
	  if( abs(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
		printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void vecGPU (float* ad, float* bd, float* cd, int n, int threads) {
	int index = threadIdx.x + blockDim.x * blockIdx.x;	
	while (index < n) {
		cd[index] += ad[index] * bd[index];
		index += threads;
	}
}
